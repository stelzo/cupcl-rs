#include "hip/hip_runtime.h"
#include "passthrough.hpp"
#include <hip/hip_runtime.h>
#include <cassert>
#include <stdint.h>
#include "math_utils.cuh"
#include <cmath>

typedef enum {
    PASSTHROUGH = 0,
    VOXELGRID = 1,
} FilterType_t;

typedef struct {
    FilterType_t type;
    //0=x,1=y,2=z
    //type PASSTHROUGH
    int dim;
    float upFilterLimits;
    float downFilterLimits;
    bool limitsNegative;
    //type VOXELGRID
    float voxelX;
    float voxelY;
    float voxelZ;

} FilterParam_t;


class cudaFilter
{
public:
    cudaFilter(hipStream_t stream = 0);
    ~cudaFilter(void);
    /*
    Input:
        source: data pointer for points cloud
        nCount: count of points in cloud_in
    Output:
        output: data pointer which has points filtered by CUDA
        countLeft: count of points in output
    */
    int set(FilterParam_t param);
    int filter(void *output, unsigned int *countLeft, void *source, unsigned int nCount);

    void *m_handle = NULL;
};

__forceinline__ __device__
bool inside_range(float3 p, float min_dist, float max_dist)
{
    float dist = dot_float3(p, p);
    return dist >= min_dist * min_dist && dist <= max_dist * max_dist;
}

__forceinline__ __device__
bool within_intensity(float intensity, float min_intensity, float max_intensity)
{
    return intensity >= min_intensity && intensity <= max_intensity;
}

__forceinline__ __device__
bool inside_box(float3 p, float3 min, float3 max)
{
    return p.x >= min.x && p.x <= max.x &&
           p.y >= min.y && p.y <= max.y &&
           p.z >= min.z && p.z <= max.z;
}

__forceinline__ __device__
bool inside_horizontal_fov(float3 p, float fov_right, float fov_left, float2 forward)
{
    float angle = angle2d(forward, make_float2(p.x, p.y));
    float fov_angle = safe_angle(fov_right - fov_left);
    float local_angle = safe_angle(angle - fov_left);
    return local_angle <= fov_angle;
}

__forceinline__ __device__
bool point_inside_polygon_winding_number(float3 p, float2* polygon, int polygon_size)
{
    uint8_t winding_number = 0;
    for (int i = 0; i < polygon_size; i++)
    {
        float2 v1 = polygon[i];
        float2 v2 = polygon[(i + 1) % polygon_size];

        bool cont = (p.y < min(v1.y, v2.y) || p.y > max(v1.y, v2.y)) ||
                    (p.x > max(v1.x, v2.x)) ||
                    (v1.y == v2.y);

        if (cont) continue;

        const float y_slope = (v2.x - v1.x) / (v2.y - v1.y);
        const float x_intercept = (p.y - v1.y) * y_slope + v1.x;
        if (v1.x == v2.x || p.x <= x_intercept)
        {
            winding_number += v2.y - v1.y > 0 ? 1 : -1;
        }
    }

    return winding_number != 0;
}

struct PassthroughParams {
    uint32_t point_step;
    float min_dist;
    float max_dist;
    float3 min;
    float3 max;
    bool invert_bounding_box;
    bool invert_distance;
    float4 rotation;
    float3 translation;
    float fov_right;
    float fov_left;
    float2 forward;
    bool enable_horizontal_fov;
    bool invert_fov;
    float2* polygon;
    int polygon_size;
    bool invert_polygon;
    float min_intensity;
    float max_intensity;
    bool invert_intensity;
};

/**
 * Filter out points that are not within the specified range.
 * The range is specified as a minimum and maximum distance from the origin.
 * Input cloud can be arbitrary point step size but output will be float4. 
 */
__global__
void krnl_passthrough_filter(
    void* cloud,
    uint32_t num_points, 
    float* cloud_filtered,
    uint32_t* num_points_filtered,
    PassthroughParams params
    )
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    float4 p = *((float4*) ((int8_t*)cloud + (params.point_step * idx))); // int8_t* to avoid pointer arithmetic but we go byte steps
    float3 p_t = transform_point(make_float3(p.x, p.y, p.z), params.rotation, params.translation);

    bool is_inside_range = inside_range(p_t, params.min_dist, params.max_dist);
    if (params.invert_distance) is_inside_range = !is_inside_range;

    bool is_within_intensity = within_intensity(p.w, params.min_intensity, params.max_intensity);
    if(params.invert_intensity) is_within_intensity = !is_within_intensity;
  
    bool is_inside_box = inside_box(p_t, params.min, params.max);
    if (params.invert_bounding_box) is_inside_box = !is_inside_box;

    bool is_inside_fov = inside_horizontal_fov(p_t, params.fov_right, params.fov_left, params.forward);
    if (params.invert_fov) is_inside_fov = !is_inside_fov;


    bool is_inside_polygon = point_inside_polygon_winding_number(p_t, params.polygon, params.polygon_size);
    if (params.invert_polygon) is_inside_polygon = !is_inside_polygon;

    // compute as much as possible before branching or atomicAdd so the kernels run in simd
    if (!is_inside_range || !is_within_intensity || !is_inside_box ||
        (params.enable_horizontal_fov && !is_inside_fov) ||
        (params.polygon_size != 0 && !is_inside_polygon)) {
        return;
    }

    uint32_t idx_filtered = atomicAdd(num_points_filtered, 1);
    cloud_filtered[idx_filtered * 4 + 0] = p.x;
    cloud_filtered[idx_filtered * 4 + 1] = p.y;
    cloud_filtered[idx_filtered * 4 + 2] = p.z;
    cloud_filtered[idx_filtered * 4 + 3] = p.w;
}

extern "C"
{
void cupcl_passthrough_filter(
    void* stream,
    void* cloud,
    uint32_t num_points, 
    uint32_t point_step, 
    float min_dist,
    float max_dist,
    float min_x,
    float min_y,
    float min_z,
    float max_x,
    float max_y,
    float max_z,
    bool invert_bounding_box,
    bool invert_distance,
    float* cloud_filtered,
    uint32_t* num_points_filtered,
    float rotation_x,
    float rotation_y,
    float rotation_z,
    float rotation_w,
    float translation_x,
    float translation_y,
    float translation_z,
    float fov_right,
    float fov_left,
    float forward_x,
    float forward_y,
    bool enable_horizontal_fov,
    bool invert_fov,
    float* polygon,
    int polygon_size,
    bool invert_polygon,
    float min_intensity,
    float max_intensity,
    bool invert_intensity)
{
    constexpr size_t THREADS_PER_BLOCK = 256;
    size_t BLOCKS = (num_points + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipStream_t s = (hipStream_t)stream;

    struct PassthroughParams params;
    params.point_step = point_step;
    params.min_dist = min_dist;
    params.max_dist = max_dist;
    params.min = make_float3(min_x, min_y, min_z);
    params.max = make_float3(max_x, max_y, max_z);
    params.invert_bounding_box = invert_bounding_box;
    params.invert_distance = invert_distance;
    params.rotation = make_float4(rotation_x, rotation_y, rotation_z, rotation_w);
    params.translation = make_float3(translation_x, translation_y, translation_z);
    params.fov_right = safe_angle(fov_right);
    params.fov_left = safe_angle(fov_left);
    params.forward = make_float2(forward_x, forward_y);
    params.enable_horizontal_fov = enable_horizontal_fov;
    params.invert_fov = invert_fov;
    params.polygon = (float2*)polygon;
    params.polygon_size = polygon_size;
    params.invert_polygon = invert_polygon;
    params.min_intensity = min_intensity;
    params.max_intensity = max_intensity;
    params.invert_intensity = invert_intensity;

    krnl_passthrough_filter<<<BLOCKS, THREADS_PER_BLOCK, 0, s>>>(
        cloud,
        num_points,
        cloud_filtered,
        num_points_filtered,
        params
    );
    hipStreamSynchronize(s);
}


void* cupcl_init_voxel_filter(void* stream, float voxel_size_x, float voxel_size_y, float voxel_size_z)
{
    hipStream_t s = (hipStream_t)stream;
    cudaFilter* filter = new cudaFilter(s);
    FilterParam_t param;
    param.type = VOXELGRID;
    param.voxelX = voxel_size_x;
    param.voxelY = voxel_size_y;
    param.voxelZ = voxel_size_z;
    filter->set(param);
    return 0;
}

uint32_t cupcl_voxel_filter(void* filter_instance, void* stream, float* input, uint32_t input_n, float* filtered)
{
    hipStream_t s = (hipStream_t)stream;
    cudaFilter* f = (cudaFilter*)filter_instance;
    uint32_t filtered_n = 0;
    int32_t ret = f->filter(filtered, &filtered_n, input, input_n);
    assert(ret == 0);
    hipStreamSynchronize(s);
    return filtered_n;
}

void cupcl_free_voxel_filter(void* filter_instance)
{
    if (filter_instance == NULL)
        return;
    cudaFilter* f = (cudaFilter*)filter_instance;
    delete f;
}
}